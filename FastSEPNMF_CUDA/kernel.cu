#include "hip/hip_runtime.h"

#include "kernel.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <sys/time.h>
#include <sys/resource.h>


__global__ void actualizacion(float *v_c, float *image_c, int bands, float *normM_c, long int image_size) { 
	int k;
	float faux = 0;
	int j =  blockIdx.x * blockDim.x + threadIdx.x;
	if (j < image_size){
		faux = 0;
		for(k = 0; k < bands; k++){
			faux += v_c[k] * image_c[k*image_size + j];
		}
		normM_c[j] -= faux * faux;
	}

}


__global__ void normalizacion(float *image_c, int bands, long int image_size, float *normM_c) { 
	long int j, i;
	float norm_val = 0, aux = 0, pixel = 0;
	
	i =  blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < image_size){
        for(j = 0; j < bands; j++){
           norm_val += image_c[j*image_size + i]; 
        } 
		
		norm_val = 1.0/(norm_val + 1.0e-16);
	
		for(j = 0; j < bands; j++){
            pixel = image_c[j*image_size  + i] * norm_val;
            image_c[j*image_size + i] = pixel;
            aux += pixel * pixel;
        }
        normM_c[i] = aux;
	}

}

__global__ void calculateNormM(float *image_c, int bands, long int image_size, float *normM_c) { 
	int k;
	int j =  blockIdx.x * blockDim.x + threadIdx.x;
	
	if (j < image_size){
		for(k = 0; k < bands; k++){
				normM_c[j] += image_c[k*image_size + j] * image_c[k*image_size + j]; 
			}
	}

}



void checkCUDAError(const char *mensaje, hipError_t error){
	
	if(error != hipSuccess){
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje);
	
	}
}

void reservarMemoria( int bands, long int image_size, float **v_c, float **image_c, float **normM_c, float **image, float **normM, float **v){
	hipError_t error;
	
	error = hipMalloc(v_c, bands*sizeof(float));
	checkCUDAError("ERROR EN hipMalloc de v_c", error);
	
	error = hipMalloc(image_c, bands*image_size*sizeof(float));
	checkCUDAError("ERROR EN hipMalloc de image_c", error);
	
	error = hipMalloc(normM_c, image_size*sizeof(float));
	checkCUDAError("ERROR EN hipMalloc de normMc", error);
	
	error = hipHostAlloc(image, bands*image_size*sizeof(float), hipHostMallocDefault); 
	checkCUDAError("ERROR EN hipHostAlloc de image", error);
	
	error = hipHostAlloc(normM, image_size*sizeof(float), hipHostMallocDefault);
	checkCUDAError("ERROR EN hipHostAlloc de normM", error);
	
	error = hipHostAlloc(v, image_size*sizeof(float), hipHostMallocDefault);
	checkCUDAError("ERROR EN hipHostAlloc de v", error);

}


void liberarMemoria(float *v_c, float *image_c, float *normM_c, float *image, float *normM, float *v){
	hipError_t error;
	
	error = hipFree(v_c);
	checkCUDAError("ERROR EN hipFree de v_c", error);
	
	error = hipFree(image_c);
	checkCUDAError("ERROR EN hipFree de image_c", error);
	
	error = hipFree(normM_c);
	checkCUDAError("ERROR EN hipFree de normM_c", error);
	
	error = hipHostFree(image);
	checkCUDAError("ERROR EN hipHostFree de image", error);
	
	error = hipHostFree(normM);
	checkCUDAError("ERROR EN hipHostFree de normM", error);
	
	error = hipHostFree(v);
	checkCUDAError("ERROR EN hipHostFree de v", error);
	
}

void selectDevice(){
	int count;
	int	i, device;
	hipDeviceProp_t prop;
	hipError_t error;
	
	hipGetDeviceCount(&count);
	
	for(i = 0; i < count; ++i){
		hipGetDeviceProperties(&prop, i);
		printf("Device %d, con nombre: %s\n", i, prop.name);		
	}
	
	printf("Select a device: ");
	scanf ("%d", &device);
		
	error = hipSetDevice(device);
	checkCUDAError("ERROR EN setDevice", error);
	
	
}


void actualizarNormM(float *v, int bands, float *normM, long int image_size, int i, int rows, float *v_c, float *image_c, float *normM_c, float *t_act){
	hipError_t error;
	struct timeval t1, t2;
	float t_sec, t_usec;
		
	error = hipMemcpy(v_c, v, bands*sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError("ERROR EN cudamemcpy de v_c", error);
	
	dim3 dimBlock(1024);
	dim3 dimGrid(ceil(image_size/1024)); 
	
	gettimeofday(&t1,NULL);
	
	actualizacion<<<dimGrid,dimBlock>>>(v_c, image_c, bands, normM_c, image_size);
	checkCUDAError("ERROR EN kernel actualización", hipGetLastError());
	hipDeviceSynchronize();	
	
	gettimeofday(&t2,NULL);
	t_sec  = (float)  (t2.tv_sec - t1.tv_sec);
	t_usec = (float)  (t2.tv_usec - t1.tv_usec);
	t_act[0] = t_act[0] + t_sec + t_usec/1.0e+6;
		
	error = hipMemcpy(normM, normM_c, image_size*sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("ERROR EN hipMemcpy de normM", error);
	
	
}


void normalizeImgC(float *image, long int image_size, int bands,float *image_c, int rows, float *normM_c, float *normM, float *normM1, float *t_copia_v, float *t_normalizar, float *t_copianorm){
	hipError_t error;
	struct timeval t1, t2;
	float t_sec, t_usec;

	gettimeofday(&t1,NULL);
	error = hipMemcpy(image_c, image, bands*image_size*sizeof(float), hipMemcpyHostToDevice); // Bueno
	checkCUDAError("ERROR EN hipMemcpy de image_c", error);
	
	gettimeofday(&t2,NULL);
	t_sec  = (float)  (t2.tv_sec - t1.tv_sec);
	t_usec = (float)  (t2.tv_usec - t1.tv_usec);
	t_copia_v[0] = t_copia_v[0] + t_sec + t_usec/1.0e+6;
		
	dim3 dimBlock(1024);
	dim3 dimGrid(ceil(image_size/1024)); 

	normalizacion<<<dimGrid,dimBlock>>>(image_c, bands, image_size, normM_c);
	checkCUDAError("ERROR EN kernel normalización", hipGetLastError());
	hipDeviceSynchronize();
	
	error = hipMemcpy(normM, normM_c, image_size*sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("ERROR EN cudamemcpy de normM_c", error);


}

void calculateNormM(float *image, float *normM, float *normM1, long int image_size, int bands, int rows, float *image_c, float *normM_c){
	hipError_t error;
	
	error = hipMemcpy(image_c, image, bands*image_size*sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError("ERROR EN hipMemcpy de image_c", error);
	
	dim3 dimBlock(1024);
	dim3 dimGrid(ceil(image_size/1024)); 
	
	calculateNormM<<<dimGrid,dimBlock>>>(image_c, bands, image_size, normM_c);
	checkCUDAError("ERROR EN kernel calculateNormM", hipGetLastError());
	hipDeviceSynchronize();
	
	error = hipMemcpy(normM, normM_c, image_size*sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("ERROR EN cudamemcpy de normM_c", error);

}





