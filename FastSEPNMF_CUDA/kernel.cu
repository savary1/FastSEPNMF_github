#include "hip/hip_runtime.h"

#include "kernel.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <sys/time.h>
#include <sys/resource.h>

#define DIM 224

__global__ void maxVal(float *normM_c, long int image_size, float *d_projections){
	__shared__ float val[1024];
	
	unsigned int tid = threadIdx.x;
    unsigned int id = blockIdx.x * blockDim.x*2 + threadIdx.x;
				
	if(id < image_size){
		if((id+blockDim.x) >= image_size){
			val[tid] = normM_c[id];
		}
		else{
			if(normM_c[id]>normM_c[id + blockDim.x]){
				val[tid]=normM_c[id];
			}
			else{
				val[tid]=normM_c[id + blockDim.x];
			}
		}
	}
	else{
		val[tid] = -1;
	}
	
	__syncthreads();
	
	for (unsigned int s = blockDim.x / 2; s > 0; s>>=1){
		if (tid < s){
			if(val[tid]<=val[tid+s]){
				val[tid]=val[tid+s];
			}
		}
		__syncthreads();
	}
	d_projections[blockIdx.x]=val[0];
	
	__syncthreads();
}



__global__ void maxValExtract(float *normM_c, float *normM1_c, long int image_size, float *d_projections, int *d_index, float a){ 
	__shared__ int pos[2048];
	__shared__ float val[2048];
	
	unsigned int tid = threadIdx.x;
    unsigned int id = blockIdx.x*2 * blockDim.x + threadIdx.x;
	float faux, faux2;
	
	faux = ((a - normM_c[id])/a);
	faux2 = ((a - normM_c[id + blockDim.x])/a);
		
	
	if(id < image_size && faux <= 1.0e-6){
		val[tid] = normM1_c[id];
		pos[tid] = id;
		
	}
	else{
		val[tid] = -1;
	}
	
	if(id + blockDim.x < image_size && faux2 <= 1.0e-6){
		val[tid + blockDim.x] = normM1_c[id + blockDim.x];
		pos[tid + blockDim.x] = id + blockDim.x;		
	}
	else{
		val[tid + blockDim.x] = -1;
	}
	__syncthreads();
	
	for (unsigned int s = blockDim.x; s > 0; s>>=1){
		if (tid < s){
			if(val[tid]<=val[tid+s]){
				val[tid] = val[tid+s];
				pos[tid] = pos[tid+s];
			}			
		}
		__syncthreads();
	}
	
	d_projections[blockIdx.x]=val[0];
	d_index[blockIdx.x]=(int)pos[0];


	__syncthreads();	
}


__global__ void actualizacion(float *v_c, float *image_c, int bands, float *normM_c, long int image_size) { 
	__shared__ float block_v[DIM];
	int k, i;
	float faux = 0;
	int j =  blockIdx.x * blockDim.x + threadIdx.x;
		
		
	if(blockDim.x < bands){
		for(i = threadIdx.x; i < bands; i += blockDim.x){
			block_v[i] = v_c[i];
		}
	}
	else{
		if(threadIdx.x < bands){
			block_v[threadIdx.x] = v_c[threadIdx.x];
		}
	}
	__syncthreads();
	
	if (j < image_size){	
		faux = 0;
		for(k = 0; k < bands; k++){
			faux += block_v[k] * image_c[k*image_size + j];
		}
		normM_c[j] -= faux * faux;
	}
}


__global__ void normalizacion(float *image_c, int bands, long int image_size, float *normM_c, float *normM1_c) { 
	long int j, i;
	float norm_val = 0, aux = 0, pixel = 0;
	
	i =  blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < image_size){
        for(j = 0; j < bands; j++){
           norm_val += image_c[j*image_size + i]; 
        } 
		
		norm_val = 1.0/(norm_val + 1.0e-16);
	
		for(j = 0; j < bands; j++){
            pixel = image_c[j*image_size  + i] * norm_val;
            image_c[j*image_size + i] = pixel;
            aux += pixel * pixel;
        }
        normM_c[i] = aux;
		normM1_c[i] = aux;
	}

}

__global__ void calculateNormM(float *image_c, int bands, long int image_size, float *normM_c, float *normM1_c) { 
	int k;
	int j =  blockIdx.x * blockDim.x + threadIdx.x;
	
	if (j < image_size){
		for(k = 0; k < bands; k++){
				normM_c[j] += image_c[k*image_size + j] * image_c[k*image_size + j]; 
				normM1_c[j] += image_c[k*image_size + j] * image_c[k*image_size + j];
			}
	}

}

void checkCUDAError(const char *mensaje, hipError_t error){
	
	if(error != hipSuccess){
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje);
	
	}
}

void reservarMemoria(int bands,long int image_size, float **v_c, float **image_c, float **normM_c, float **normM1_c, float **image, float **v, float **d_projections, int **d_index, float **h_projections, int **h_index, int globalSize_reduction){	
	hipError_t error;
	
	error = hipMalloc(v_c, bands*sizeof(float));
	checkCUDAError("ERROR EN hipMalloc de v_c", error);
	
	error = hipMalloc(image_c, bands*image_size*sizeof(float));
	checkCUDAError("ERROR EN hipMalloc de image_c", error);
	
	error = hipMalloc(normM_c, image_size*sizeof(float));
	checkCUDAError("ERROR EN hipMalloc de normMc", error);
	
	error = hipMalloc(normM1_c, image_size*sizeof(float));
	checkCUDAError("ERROR EN hipMalloc de normM1_c", error);
	
	error = hipHostAlloc(image, bands*image_size*sizeof(float), hipHostMallocDefault); 
	checkCUDAError("ERROR EN hipHostAlloc de image", error);
	
	error = hipHostAlloc(v, image_size*sizeof(float), hipHostMallocDefault);
	checkCUDAError("ERROR EN hipHostAlloc de v", error);
	
	error = hipMalloc(d_projections, globalSize_reduction*sizeof(float));
	checkCUDAError("ERROR EN hipMalloc de d_projections", error);
	
	error = hipMalloc(d_index, globalSize_reduction*sizeof(int));
	checkCUDAError("ERROR EN hipMalloc de d_projections", error);
	
	error = hipHostAlloc(h_projections, globalSize_reduction*sizeof(float), hipHostMallocDefault);
	checkCUDAError("ERROR EN hipHostAlloc de d_projections", error);
	
	error = hipHostAlloc(h_index, globalSize_reduction*sizeof(int), hipHostMallocDefault);
	checkCUDAError("ERROR EN hipHostAlloc de d_projections", error);
	

}


void liberarMemoria(float *v_c, float *image_c, float *normM_c, float *image, float *normM1_c, float *v, float *d_projections, int *d_index, float *h_projections, int *h_index){
	hipError_t error;
	
	error = hipFree(v_c);
	checkCUDAError("ERROR EN hipFree de v_c", error);
	
	error = hipFree(image_c);
	checkCUDAError("ERROR EN hipFree de image_c", error);
	
	error = hipFree(normM_c);
	checkCUDAError("ERROR EN hipFree de normM_c", error);
	
	error = hipHostFree(image);
	checkCUDAError("ERROR EN hipHostFree de image", error);
	
	error = hipFree(normM1_c);
	checkCUDAError("ERROR EN hipHostFree de normM1_c", error);
	
	error = hipHostFree(v);
	checkCUDAError("ERROR EN hipHostFree de v", error);
	
	error = hipFree(d_projections);
	checkCUDAError("ERROR EN hipFree de d_projections", error);

	error = hipFree(d_index);
	checkCUDAError("ERROR EN hipFree de d_index", error);	
	
	error = hipHostFree(h_projections);
	checkCUDAError("ERROR EN hipHostFree de h_projections", error);
	
	error = hipHostFree(h_index);
	checkCUDAError("ERROR EN hipHostFree de h_index", error);	
	
	
}

void selectDevice(){
	int count;
	int	i , device;
	hipDeviceProp_t prop;
	hipError_t error;
	
	hipGetDeviceCount(&count);
	
	for(i = 0; i < count; ++i){
		hipGetDeviceProperties(&prop, i);
		printf("Device %d, con nombre: %s\n", i, prop.name);		
	}
	
	printf("Select a device: ");
	scanf ("%d", &device);
		
	error = hipSetDevice(device);
	checkCUDAError("ERROR EN setDevice", error);
	
	
}


void actualizarNormM(float *v, int bands, long int image_size, int i, int rows, float *v_c, float *image_c, float *normM_c){
	hipError_t error;
	int val = ceil((double)image_size/1024);
		
	error = hipMemcpy(v_c, v, bands*sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError("ERROR EN cudamemcpy de v_c", error);
	
	dim3 dimBlock(1024);
	dim3 dimGrid(val); 
	
	
	actualizacion<<<dimGrid,dimBlock>>>(v_c, image_c, bands, normM_c, image_size);
	checkCUDAError("ERROR EN kernel actualización", hipGetLastError());
	hipDeviceSynchronize();	
	
	
	
}

void normalizeImgC(float *image, long int image_size, int bands,float *image_c, int rows, float *normM_c, float *normM1_c){
	hipError_t error;
	int val = ceil((double)image_size/1024);

	error = hipMemcpy(image_c, image, bands*image_size*sizeof(float), hipMemcpyHostToDevice); 
	checkCUDAError("ERROR EN hipMemcpy de image_c", error);
	
		
	dim3 dimBlock(1024);
	dim3 dimGrid(val); 

	normalizacion<<<dimGrid,dimBlock>>>(image_c, bands, image_size, normM_c, normM1_c);
	checkCUDAError("ERROR EN kernel normalización", hipGetLastError());
	hipDeviceSynchronize();


}

void calculateNormM(float *image, long int image_size, int bands, int rows, float *image_c, float *normM_c, float *normM1_c){
	hipError_t error;
	int val = ceil((double)image_size/1024);
	
	error = hipMemcpy(image_c, image, bands*image_size*sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError("ERROR EN hipMemcpy de image_c", error);
	
	dim3 dimBlock(1024);
	dim3 dimGrid(val); 
	
	calculateNormM<<<dimGrid,dimBlock>>>(image_c, bands, image_size, normM_c, normM1_c);
	checkCUDAError("ERROR EN kernel calculateNormM", hipGetLastError());
	hipDeviceSynchronize();
	

}


void calculateMaxVal(int image_size, float *normM_c, float *d_projections, float *h_projections){
	
	int val = ceil((double)image_size/2/1024);
	hipError_t error;
	
	dim3 dimBlock(1024);
	dim3 dimGrid(val); 

	
	maxVal<<<dimGrid,dimBlock>>>(normM_c, image_size, d_projections);
	checkCUDAError("ERROR EN kernel maxVal", hipGetLastError());
	hipDeviceSynchronize();	
	
	error = hipMemcpy(h_projections, d_projections, val*sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("ERROR EN hipMemcpy de h_projections", error);


}


void calculateMaxValExtract_2(int image_size, float *normM_c, float *normM1_c, float *d_projections, float *h_projections, int *d_index, int *h_index, float a){
	
	hipError_t error;
	int val = ceil((double)image_size/2/1024);
	
	dim3 dimBlock(1024);
	dim3 dimGrid(val); 
	
	maxValExtract<<<dimGrid,dimBlock>>>(normM_c, normM1_c, image_size, d_projections, d_index, a);
	checkCUDAError("ERROR EN kernel maxValExtract", hipGetLastError());
	hipDeviceSynchronize();	
	
	error = hipMemcpy(h_projections, d_projections, val*sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("ERROR EN hipMemcpy de h_projections", error);
	
	error = hipMemcpy(h_index, d_index, val*sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("ERROR EN hipMemcpy de h_index", error);

}


